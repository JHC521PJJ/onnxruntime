#include "hip/hip_runtime.h"
#include "resultTransformate.cuh"
#include <vector>
#include <hip/hip_runtime_api.h>
#include <stdio.h>

static constexpr int channel = 384;
static constexpr int out_size = 56;


__global__ void squareDifferenceKernel(float* d_teacher, float* d_student, float* d_autoencoder, 
    float* d_map_st, float* d_map_ae, const int size) {

    unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;

    if(idx < size) {
        d_map_st[idx] = d_teacher[idx] - d_student[idx];
        d_map_ae[idx] = d_autoencoder[idx] - d_student[channel * 56 * 56 + idx];
        d_map_st[idx] *= d_map_st[idx]; 
        d_map_ae[idx] *= d_map_ae[idx]; 
    }
}

__global__ void meanKernel(float* d_map_st, float* d_map_ae, 
    float* d_mean_st, float* d_mean_ae, const int size) {
        
    unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
    float temp_st = 0.0f;
    float temp_ae = 0.0f;

    if(idx < size) {
        for(int i = 0; i < channel; ++i) {
            temp_st += d_map_st[idx + i * 56 * 56];
            temp_ae += d_map_ae[idx + i * 56 * 56];
        }
        d_mean_st[idx] = temp_st / channel;
        d_mean_ae[idx] = temp_ae / channel;
    }
}

__global__ void combineKernel(float* d_mean_st, float* d_mean_ae, float* d_combine,
    float* d_st_start_quantiles,
    float* d_st_end_quantiles,
    float* d_ae_start_quantiles,
    float* d_ae_end_quantiles,
    const int size) {
    
    unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;

    if(idx < size) {
        d_mean_st[idx] = 0.1f * (d_mean_st[idx] - *d_st_start_quantiles) / (*d_st_end_quantiles - *d_st_start_quantiles);
        d_mean_ae[idx] = 0.1f * (d_mean_ae[idx] - *d_ae_start_quantiles) / (*d_ae_end_quantiles - *d_ae_start_quantiles);
        d_combine[idx] = 0.5f * d_mean_st[idx] + 0.5f * d_mean_ae[idx];
    }
}

void resultTransformate(const std::vector<float>& t_output, 
    const std::vector<float>& s_output,
    const std::vector<float>& ae_output,
    float q_st_start_quantiles,
    float q_st_end_quantiles,
    float q_ae_start_quantiles,
    float q_ae_end_quantiles,
    const int device_id,
    std::vector<float>& vec_combine) {

    hipSetDevice(device_id);

    float* d_teacher; float* d_student; float* d_autoencoder;
    float* d_map_st; float* d_map_ae;
    float* d_mean_st; float* d_mean_ae;
    float* d_combine;
    float* d_st_start_quantiles;
    float* d_st_end_quantiles;
    float* d_ae_start_quantiles;
    float* d_ae_end_quantiles; 

    hipMalloc((void **) &d_teacher, sizeof(float) * channel * out_size * out_size);
    hipMalloc((void **) &d_student, sizeof(float) * channel * 2 * out_size * out_size);
    hipMalloc((void **) &d_autoencoder, sizeof(float) * channel * out_size * out_size);
    hipMalloc((void **) &d_map_st, sizeof(float) * channel * out_size * out_size);
    hipMalloc((void **) &d_map_ae, sizeof(float) * channel * out_size * out_size);
    hipMalloc((void **) &d_mean_st, sizeof(float) * out_size * out_size);
    hipMalloc((void **) &d_mean_ae, sizeof(float) * out_size * out_size);
    hipMalloc((void **) &d_combine, sizeof(float) * out_size * out_size);
    hipMalloc((void **) &d_st_start_quantiles, sizeof(float));
    hipMalloc((void **) &d_st_end_quantiles, sizeof(float));
    hipMalloc((void **) &d_ae_start_quantiles, sizeof(float));
    hipMalloc((void **) &d_ae_end_quantiles, sizeof(float));

    hipMemcpy(d_teacher, t_output.data(), sizeof(float) * channel * out_size * out_size, hipMemcpyHostToDevice);
    hipMemcpy(d_student, s_output.data(), sizeof(float) * channel * 2 *  out_size * out_size, hipMemcpyHostToDevice);
    hipMemcpy(d_autoencoder, ae_output.data(), sizeof(float) * channel * out_size * out_size, hipMemcpyHostToDevice);
    hipMemcpy(d_st_start_quantiles, &q_st_start_quantiles, sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_st_end_quantiles, &q_st_end_quantiles, sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_ae_start_quantiles, &q_ae_start_quantiles, sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_ae_end_quantiles, &q_ae_end_quantiles, sizeof(float), hipMemcpyHostToDevice);

    int size = channel * out_size * out_size;
    unsigned int block_size = 16 * 16;
    unsigned int grid_size = (size + block_size - 1) / block_size;
    dim3 grid_dim(grid_size);
    dim3 block_dim(block_size);
    squareDifferenceKernel<<<grid_dim, block_dim>>>(d_teacher, d_student, d_autoencoder, d_map_st, d_map_ae, size);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA Error: %s\n", hipGetErrorString(err));
    }
    hipDeviceSynchronize();

    size = out_size * out_size;
    grid_size = (size + block_size - 1) / block_size;
    meanKernel<<<grid_size, block_dim>>>(d_map_st, d_map_ae, d_mean_st, d_mean_ae, size);
    if (err != hipSuccess) {
        printf("CUDA Error: %s\n", hipGetErrorString(err));
    }
    hipDeviceSynchronize();

    combineKernel<<<grid_size, block_dim>>>(d_mean_st, d_mean_ae, d_combine, d_st_start_quantiles, d_st_end_quantiles, d_ae_start_quantiles, d_ae_end_quantiles, size);
    if (err != hipSuccess) {
        printf("CUDA Error: %s\n", hipGetErrorString(err));
    }
    hipMemcpy(vec_combine.data(), d_combine, sizeof(float) * out_size * out_size, hipMemcpyDeviceToHost);

    hipFree(d_teacher);
    hipFree(d_student);
    hipFree(d_autoencoder);
    hipFree(d_map_st);
    hipFree(d_map_ae);
    hipFree(d_mean_st);
    hipFree(d_mean_ae);
    hipFree(d_combine);
    hipFree(d_st_start_quantiles);
    hipFree(d_st_end_quantiles);
    hipFree(d_ae_start_quantiles);
    hipFree(d_ae_end_quantiles); 
}